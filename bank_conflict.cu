#include<iostream>
#include<hip/hip_runtime.h>
#include<time.h>
using namespace std;

__global__ void kernel( unsigned long long* d_Time, int* d_a, int* foo) {
    __shared__ float MySharedMemory[8192];
    unsigned long long Time_start;
    unsigned long long Time_stop;
    Time_start = clock();
    if(threadIdx.x%2 == 0){
        d_a[0]++;
    } else{
        d_a[*foo]++;
    }
    Time_stop = clock();
    *d_Time = (Time_stop-Time_start);
}
int main( void ) {
    int *d_foo;
    int* foo = (int*) malloc(sizeof(int));
    hipMalloc((void**) &d_foo,sizeof(int));
    int reps=10;
    unsigned long long Time;
    unsigned long long* d_Time;
    hipMalloc((void**)&d_Time,sizeof(unsigned long long));
    int* a;
    int* d_a;
    hipMalloc((void**)&d_a, sizeof(int)*33);

    hipMemcpy(d_Time,&Time, sizeof(unsigned long long), hipMemcpyHostToDevice);
    hipMemcpy(d_a,a, sizeof(int)*33, hipMemcpyHostToDevice);
    *foo = 32;
    hipMemcpy(d_foo,foo,sizeof(int),hipMemcpyHostToDevice);
    kernel<<<1,32>>>(d_Time, d_a, d_foo);
    hipMemcpy(&Time,d_Time, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    cout<<"Elapsed Time with conflict:    "<<(Time)<<endl;
    *foo = 31;
    hipMemcpy(d_foo,foo,sizeof(int),hipMemcpyHostToDevice);
    kernel<<<1,32>>>(d_Time, d_a, d_foo);
    hipMemcpy(&Time,d_Time, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    cout<<"Elapsed Time without conflict: "<<(Time)<<endl;
    hipFree(d_Time);
    return 0;
}
